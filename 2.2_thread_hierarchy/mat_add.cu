
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <random>

#define N 4 

// Kernel definition
__global__ void MatAdd(float A[N][N], float B[N][N], float C[N][N])
{
    // threadIdx is a predefined thread variable

    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}

int main()
{
    // ----------------------------------------------------------------------
    // 1. Declare host (CPU) 2D arrays.
    //    Each is physically N*N floats in contiguous memory on the CPU.
    // ----------------------------------------------------------------------
    float host_A[N][N], host_B[N][N], host_C[N][N];

    // Fill host arrays with random data in [0,1].
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            // (float) cast ensures float division (rand() returns an int).
            host_A[i][j] = (float)rand() / (float)RAND_MAX;
            host_B[i][j] = (float)rand() / (float)RAND_MAX;
        }
    }

    /*
      ------------------------------------------------------------------------
      2. Declare pointers for the device (GPU) memory.

      float (*device_A)[N];
      ---------------------
      "device_A is a pointer to an array of N floats." 
      That means we can use device_A[i][j] in a kernel as if it were A[i][j].
      But physically it's still a single contiguous block of memory sized N*N.

      Similarly for device_B and device_C.
      ------------------------------------------------------------------------
    */
    float (*device_A)[N];
    float (*device_B)[N];
    float (*device_C)[N];

    /*
      ------------------------------------------------------------------------
      3. cudaMalloc(void **devPtr, size_t size)
         - devPtr: pointer to a pointer to void. In other words, a parameter
                   where cudaMalloc will write back the GPU address it allocates.
         - size: how many bytes we want on the GPU.

         We pass &device_A (address of our pointer 'device_A') so cudaMalloc
         can store the allocated device address into 'device_A'.

         Because device_A is type "pointer to an array of N floats"
         (float (*)[N]), &device_A is thus "pointer to that pointer," or
         float (**)[N]. We cast it to (void**) to match the cudaMalloc signature.

         We need N*N*sizeof(float) bytes on the GPU for an N x N array of floats.
      ------------------------------------------------------------------------
    */
    hipMalloc((void**)&device_A, N * N * sizeof(float));
    hipMalloc((void**)&device_B, N * N * sizeof(float));
    hipMalloc((void**)&device_C, N * N * sizeof(float));

    hipMemcpy(device_A, host_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_B, host_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    //                  4 x 4 
    dim3 threadsPerBlock(N,N);

    MatAdd<<<1,threadsPerBlock>>>(device_A,device_B,device_C);
    //       ^ num blocks   
          
    // 4x4 matrix, launch 4x4=16 threads, each thread does one computation



    hipDeviceSynchronize();
    hipMemcpy(host_C, device_C, N * N * sizeof(float), hipMemcpyDeviceToHost);


    // 6. Check the result
    for(int i = 0; i < N; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            // Expect 1.0 + 2.0 = 3.0
            float gt_c = host_A[i][j] + host_B[i][j];
            printf("ground truth c: %f, calculated c: %f\n", gt_c,host_C[i][j]);
            assert(fabs(host_C[i][j] - gt_c) < 1e-10);
        }
    }

    printf("Matrix add test passed!\n");

    // 7. Free device memory
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}
